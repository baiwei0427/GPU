
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void usage(char *program)
{     
        fprintf(stderr, "usage: %s memsize iters\n", program);
        fprintf(stderr, "    memsize: memory allocated in bytes (>0)\n");
        fprintf(stderr, "    iters: number of iterations (>0)\n");
}

int main(int argc, char **argv)
{
        int size, iters, i;
        void *ptr;
        float time, cumulative_malloc_time = 0.f, cumulative_free_time = 0.f;
        hipEvent_t start, stop;

        if (argc != 3) {
                usage(argv[0]);
                return EXIT_FAILURE;
        }

        size = atoi(argv[1]);
        iters = atoi(argv[2]);

        if (size <= 0 || iters <= 0) {
                usage(argv[0]);
                return EXIT_FAILURE;
        }

        hipFree(0);

        hipEventCreate(&start);
        hipEventCreate(&stop);

        for (i = 0; i < iters; i++) {
                
                hipEventRecord(start, 0);
                if (hipMalloc(&ptr, size) != hipSuccess) {
                        fprintf(stderr, "hipMalloc error\n");
                }
                hipEventRecord(stop, 0);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&time, start, stop);
                cumulative_malloc_time += time;                                                

                hipEventRecord(start, 0);                
                if (hipFree(ptr) != hipSuccess) {
                        fprintf(stderr, "hipFree error\n");                        
                }
                hipEventRecord(stop, 0);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&time, start, stop);
                cumulative_free_time += time;
        }

        printf("hipMalloc time:  %3.5f ms \n", cumulative_malloc_time / iters);
        printf("hipFree   time:  %3.5f ms \n", cumulative_free_time / iters);

        return EXIT_SUCCESS;
}