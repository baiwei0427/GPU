#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime_api.h"

void merge(int *array, int start, int mid, int end)
{       
        int left_index, right_index, global_index;
        int left_len = mid - start + 1;
        int right_len = end - mid;
        int left[left_len];
        int right[right_len];

        // initialize left array
        for (int i = 0; i < left_len; i++) {
                left[i] = array[start + i];
        }

        // initialize right array
        for (int i = 0; i < right_len; i++) {
                right[i] = array[mid + 1 + i];
        }

        // index of left array
        left_index = 0;
        // index of right array
        right_index = 0;
        // index of merged array
        global_index = start;

        while (left_index < left_len && right_index < right_len) {
                if (left[left_index] <= right[right_index]) {
                        array[global_index++] = left[left_index++];
                } else {
                        array[global_index++] = right[right_index++];                        
                }
        }

        // copy the rest of left array 
        while (left_index < left_len) {
                array[global_index++] = left[left_index++];
        } 

        // copy the rest of right array
        while (right_index < right_len) {
                array[global_index++] = right[right_index++];
        }
}

void cpu_merge_sort(int *array, int start, int end)
{
        if (start >= end)
                return;
        
        int mid = start + (end - start) / 2;
        cpu_merge_sort(array, start, mid);
        cpu_merge_sort(array, mid + 1, end);
        merge(array, start, mid, end);
}

__global__ void gpu_merge(int *d_in, int *d_out, int size, int sorted_size) 
{
        // global ID
        int gid = blockIdx.x * blockDim.x + threadIdx.x;
        // start, end of left subarray
        int left_start = gid * 2 * sorted_size;
        int left_end = min((gid * 2 + 1) * sorted_size - 1, size - 1);
        // start, end of right subarray
        int right_start = (gid * 2 + 1) * sorted_size;
        int right_end = min((gid * 2 + 2) * sorted_size - 1, size - 1);
        
        int left_index = left_start, right_index= right_start, global_index= left_start;

        while (left_index <= left_end && right_index <= right_end) {
                if (d_in[left_index] <= d_in[right_index]) {
                        d_out[global_index++] = d_in[left_index++];
                } else {
                        d_out[global_index++] = d_in[right_index++];                        
                }
        }

        while (left_index <= left_end) {
                d_out[global_index++] = d_in[left_index++];                
        }

        while (right_index <= right_end) {
                d_out[global_index++] = d_in[right_index++];                  
        }
}

void gpu_merge_sort(int *h_in, int size)
{
        int *d_in, *d_out, *tmp;

        if (size == 1)
                return;
        
	// allocate GPU memory
	checkCudaErrors(hipMalloc((void**)&d_in, size * sizeof(int)));
        checkCudaErrors(hipMalloc((void**)&d_out, size * sizeof(int)));

        // copy input from host memory to GPU memory
        hipMemcpy(d_in, h_in, size * sizeof(int), hipMemcpyHostToDevice);

        // # of threads per block
        int threads_per_block = 1024;
        int sorted_size = 1;
        // # of blocks 
        int blocks = 1;

        while (sorted_size < size) {
                // each thread can merge at most 2 * sorted_size elements
                // how many threads do we need in total?
                int threads_total = (size + 2 * sorted_size - 1) / (2 * sorted_size);
                // total # of blocks that we need
                blocks = (threads_total + threads_per_block - 1) / threads_per_block;

                gpu_merge<<<blocks, threads_per_block>>>(d_in, d_out, size, sorted_size);
                
                sorted_size *= 2;
                // exchange input and output
                tmp = d_in;
                d_in = d_out;
                d_out = tmp;
        }

        // copy output from GPU memory to host memory
        hipMemcpy(h_in, d_in, size * sizeof(int), hipMemcpyDeviceToHost);

        // free GPU memory
        hipFree(d_in);
        hipFree(d_out);
}

int main()
{
        int array_size = 1111;
        int array[array_size];
        // sort result computed by GPU
        int h_in[array_size];
        bool result;

        // initialize random number generator
        srand(time(NULL));

        printf("Input\n");
        for (int i = 0; i < array_size; i++) {
                array[i] = rand() % array_size;
                h_in[i] = array[i];
                printf("%d ", array[i]);
        }
        printf("\n");

        // merge sort on CPU
        cpu_merge_sort(array, 0, array_size - 1);
        printf("Expected Output\n");
        for (int i = 0; i < array_size; i++) {
                printf("%d ", array[i]);
        }         
        printf("\n");        
        
        // merge sort on GPU
        gpu_merge_sort(h_in, array_size);
        printf("GPU Output\n"); 
        result = true;
        for (int i = 0; i < array_size; i++) {
                printf("%d ", h_in[i]);
                if (h_in[i] != array[i]) {
                        result = false;
                }
        }        
        printf("\n");

        if (result) {
                printf("Correct\n");
        } else {
                printf("Wrong\n");
        }
        
        return 0;
}