
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hillissteele_scan(int *d_out, int *d_in, unsigned int size)
{
        extern __shared__ int s_data[];
        // thread ID inside the block
        unsigned int tid = threadIdx.x;

        // copy input from global memory to shared memory
        s_data[tid] = d_in[tid];
        __syncthreads();

        for (int offset = 1; offset < size; offset <<= 1) {
                int tmp = s_data[tid];
                if (tid >= offset) {
                        tmp += s_data[tid - offset];
                }
                __syncthreads();
                s_data[tid] = tmp;
                __syncthreads(); 
        }

        // copy output from shared memory to global memory
        d_out[tid] = s_data[tid];
}

__global__ void hillissteele_scan2(int *d_out, int *d_in, unsigned int size)
{
        extern __shared__ int s_data[];
        // thread ID inside the block
        unsigned int tid = threadIdx.x;
        int in = 0, out = 1;

        // copy input from global memory to shared memory
        // s_data actually has two arrays: in and out
        s_data[out * size + tid] = d_in[tid];
        __syncthreads();

        for (int offset = 1; offset < size; offset <<= 1) {
                // swap in and out
                in = out;
                out = 1 - in;

                s_data[out * size + tid] = s_data[in * size + tid];
                if (tid >= offset) {
                        s_data[out * size + tid] += s_data[in * size + tid - offset];
                } 

                __syncthreads();
        }

        // copy output from shared memory to global memory
        d_out[tid] = s_data[out * size + tid];
}

__global__ void blelloch_scan(int *d_out, int *d_in, unsigned int size, bool inclusive)
{
        extern __shared__ int s_data[];
        // thread ID inside the block
        unsigned int tid = threadIdx.x;

        // copy input from global memory to shared memory
        s_data[tid] = d_in[tid];
        __syncthreads();

        // up sweep
        for (int offset = 1; offset < size; offset <<= 1) {
                int tmp = s_data[tid];
                if ((tid + 1) % (offset << 1) == 0) {
                        tmp += s_data[tid - offset];
                }
                __syncthreads();
                s_data[tid] = tmp;
                __syncthreads(); 
        }

        // down sweep
        if (tid == size - 1)
                s_data[tid] = 0;
        __syncthreads();        
        
        for (int offset = size / 2; offset > 0; offset >>= 1) {
                int left, right;
                if ((tid + 1) % (offset << 1) == 0) {
                        left = s_data[tid];
                        right = s_data[tid] + s_data[tid - offset];
                }
                __syncthreads();

                if ((tid + 1) % (offset << 1) == 0) {
                        s_data[tid - offset] = left;
                        s_data[tid] = right;
                }
                __syncthreads();
        }

        // copy output from shared memory to global memory
        // by default, blelloch algorithm generates exclusive scan results 
        if (!inclusive) {
                d_out[tid] = s_data[tid];
        
        // to get inclusive results
        } else {
                if (tid < size - 1)
                        d_out[tid] = s_data[tid + 1];
                else
                        d_out[tid] = s_data[tid] + d_in[tid];
        }
}

__global__ void blelloch_scan2(int *d_out, int *d_in, unsigned int size, bool inclusive)
{
        // allocated on invocation
        extern __shared__ int s_data[];
        int tid = threadIdx.x;
        int offset = 1;

        // load input into shared memory
        s_data[2 * tid] = d_in[2 * tid]; 
        s_data[2 * tid + 1] = d_in[2 * tid + 1];

        // build sum in place up the tree
        for (int d = size >> 1; d > 0; d >>= 1) {
                __syncthreads();
                if (tid < d) {
                        int ai = offset * (2 * tid + 1) - 1;
                        int bi = offset * (2 * tid + 2) - 1;
                        s_data[bi] += s_data[ai];
                }
                offset <<= 1;
        }

        // clear the last element
        if (tid == 0) { 
                s_data[size - 1] = 0; 
        } 

        // traverse down tree & build scan        
        for (int d = 1; d < size; d <<= 1) {
                offset >>= 1;
                __syncthreads();
                if (tid < d) {
                        int ai = offset * (2 * tid + 1) - 1;
                        int bi = offset * (2 * tid + 2)-1;
                        int t = s_data[ai];
                        s_data[ai] = s_data[bi];
                        s_data[bi] += t;
                }
        }
        
        __syncthreads();

        // write results to device memory
        if (!inclusive) {
                d_out[2 * tid] = s_data[2 * tid]; 
                d_out[2 * tid + 1] = s_data[2 * tid + 1];
        } else {
                d_out[2 * tid] = s_data[2 * tid + 1];
                if (2 * tid + 2 < size) {
                        d_out[2 * tid + 1] = s_data[2 * tid + 2];
                } else {
                        d_out[2 * tid + 1] = s_data[2 * tid + 1] + d_in[size - 1];
                }
        }
}

int main()
{
        const int iters = 1000;
        const int array_size = 1 << 10;
        int h_in[array_size], h_out[array_size], scan_result[array_size];
        int *d_in, *d_out;

        for (int i = 0; i < array_size; i++) {
                h_in[i] = i + 1;
                // calculate expected inclusive scan result
                if (i == 0) {
                        scan_result[i] = h_in[i];
                } else {
                        scan_result[i] = scan_result[i - 1] + h_in[i];
                }
        }

        // allocate GPU memory
        if (hipMalloc((void**) &d_in, array_size * sizeof(int)) != hipSuccess
         || hipMalloc((void**) &d_out, array_size * sizeof(int)) != hipSuccess)
                goto out;
        
        // copy the input array from the host memory to the GPU memory
        hipMemcpy(d_in, h_in, array_size * sizeof(int), hipMemcpyHostToDevice);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);
        for (int i = 0; i < iters; i++) {
                //hillissteele_scan<<<1, array_size, array_size * sizeof(int)>>>(d_out, d_in, array_size);
                hillissteele_scan2<<<1, array_size, 2 * array_size * sizeof(int)>>>(d_out, d_in, array_size);
                //blelloch_scan<<<1, array_size, array_size * sizeof(int)>>>(d_out, d_in, array_size, true);
                //blelloch_scan2<<<1, array_size / 2, array_size * sizeof(int)>>>(d_out, d_in, array_size, true);
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float elapsed_time;
        hipEventElapsedTime(&elapsed_time, start, stop);    
        elapsed_time /= iters;      

        printf("Average time elapsed: %f ms\n", elapsed_time);

        // copy the result from the GPU memory to the host memory
        hipMemcpy(h_out, d_out, array_size * sizeof(int), hipMemcpyDeviceToHost);


        for (int i = 0; i < array_size; i++) {
                //printf("%d ", h_out[i]);
                if (h_out[i] != scan_result[i]) {
                        printf("Wrong result\n");
                        goto out;
                }
        }

        /*for (int i = 0; i < array_size; i++) {
                printf("%d ", h_out[i]);
        }

        printf("\n");*/

        printf("Correct result\n");
out:
        hipFree(d_in);
        hipFree(d_out);        
        return 0;
}