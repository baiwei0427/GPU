
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hillissteele_scan(int *d_out, int *d_in, unsigned int size)
{
        extern __shared__ int s_data[];
        // thread ID inside the block
        unsigned int tid = threadIdx.x;

        // copy input from global memory to shared memory
        s_data[tid] = d_in[tid];
        __syncthreads();

        for (int offset = 1; offset < size; offset <<= 1) {
                int tmp = s_data[tid];
                if (tid >= offset) {
                        tmp += s_data[tid - offset];
                }
                __syncthreads();
                s_data[tid] = tmp;
                __syncthreads(); 
        }

        // copy output from shared memory to global memory
        d_out[tid] = s_data[tid];
}

__global__ void blelloch_scan(int *d_out, int *d_in, unsigned int size, bool inclusive)
{
        extern __shared__ int s_data[];
        // thread ID inside the block
        unsigned int tid = threadIdx.x;

        // copy input from global memory to shared memory
        s_data[tid] = d_in[tid];
        __syncthreads();

        // up sweep
        for (int offset = 1; offset < size; offset <<= 1) {
                int tmp = s_data[tid];
                if ((tid + 1) % (offset << 1) == 0) {
                        tmp += s_data[tid - offset];
                }
                __syncthreads();
                s_data[tid] = tmp;
                __syncthreads(); 
        }

        // down sweep
        if (tid == size - 1)
                s_data[tid] = 0;
        __syncthreads();        
        
        for (int offset = size / 2; offset > 0; offset >>= 1) {
                int left, right;
                if ((tid + 1) % (offset << 1) == 0) {
                        left = s_data[tid];
                        right = s_data[tid] + s_data[tid - offset];
                }
                __syncthreads();

                if ((tid + 1) % (offset << 1) == 0) {
                        s_data[tid - offset] = left;
                        s_data[tid] = right;
                }
                __syncthreads();
        }

        // copy output from shared memory to global memory
        // by default, blelloch algorithm generates exclusive scan results 
        if (!inclusive) {
                d_out[tid] = s_data[tid];
        
        // to get inclusive results
        } else {
                if (tid < size - 1)
                        d_out[tid] = s_data[tid + 1];
                else
                        d_out[tid] = s_data[tid] + d_in[tid];
        }
}

int main()
{
        const int iters = 1000;
        const int array_size = 1 << 10;
        int h_in[array_size], h_out[array_size], scan_result[array_size];
        int *d_in, *d_out;

        for (int i = 0; i < array_size; i++) {
                h_in[i] = i + 1;
                // calculate expected inclusive scan result
                if (i == 0) {
                        scan_result[i] = h_in[i];
                } else {
                        scan_result[i] = scan_result[i - 1] + h_in[i];
                }
        }

        // allocate GPU memory
        if (hipMalloc((void**) &d_in, array_size * sizeof(int)) != hipSuccess
         || hipMalloc((void**) &d_out, array_size * sizeof(int)) != hipSuccess)
                goto out;
        
        // copy the input array from the host memory to the GPU memory
        hipMemcpy(d_in, h_in, array_size * sizeof(int), hipMemcpyHostToDevice);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);
        for (int i = 0; i < iters; i++) {
                //hillissteele_scan<<<1, array_size, array_size * sizeof(int)>>>(d_out, d_in, array_size);
                blelloch_scan<<<1, array_size, array_size * sizeof(int)>>>(d_out, d_in, array_size, true);
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float elapsed_time;
        hipEventElapsedTime(&elapsed_time, start, stop);    
        elapsed_time /= iters;      

        printf("Average time elapsed: %f ms\n", elapsed_time);

        // copy the result from the GPU memory to the host memory
        hipMemcpy(h_out, d_out, array_size * sizeof(int), hipMemcpyDeviceToHost);


        for (int i = 0; i < array_size; i++) {
                if (h_out[i] != scan_result[i]) {
                        printf("Wrong result\n");
                        goto out;
                }
        }

        printf("Correct result\n");

        /*for (int i = 0; i < array_size; i++) {
                printf("%d ", h_out[i]);
        }

        printf("\n");*/
out:
        hipFree(d_in);
        hipFree(d_out);        
        return 0;
}