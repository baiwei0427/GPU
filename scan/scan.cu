
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hillissteele_scan(int *d_out, int *d_in, unsigned int size)
{
        extern __shared__ int s_data[];
        // thread ID inside the block
        unsigned int tid = threadIdx.x;

        // copy input from global memory to shared memory
        s_data[tid] = d_in[tid];
        __syncthreads();

        for (int offset = 1; offset < size; offset <<= 1) {
                int tmp = s_data[tid];
                if (tid >= offset) {
                        tmp += s_data[tid - offset];
                }
                __syncthreads();
                s_data[tid] = tmp;
                __syncthreads(); 
        }

        // copy output from shared memory to global memory
        d_out[tid] = s_data[tid];
}

int main()
{
        const int array_size = 1 << 10;
        int h_in[array_size], h_out[array_size];
        int *d_in, *d_out;

        for (int i = 0; i < array_size; i++) {
                h_in[i] = i;
        }

        // allocate GPU memory
        if (hipMalloc((void**) &d_in, array_size * sizeof(int)) != hipSuccess
         || hipMalloc((void**) &d_out, array_size * sizeof(int)) != hipSuccess)
                goto out;
        
        // copy the input array from the host memory to the GPU memory
        hipMemcpy(d_in, h_in, array_size * sizeof(int), hipMemcpyHostToDevice);

        hillissteele_scan<<<1, array_size, array_size * sizeof(int)>>>(d_out, d_in, array_size);

        // copy the result from the GPU memory to the host memory
        hipMemcpy(h_out, d_out, array_size * sizeof(int), hipMemcpyDeviceToHost);

        for (int i = 0; i < array_size; i++) {
                printf("%d ", h_out[i]);
        }
        printf("\n");
out:
        hipFree(d_in);
        hipFree(d_out);        
        return 0;
}