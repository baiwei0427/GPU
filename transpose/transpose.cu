#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include "hip/hip_runtime_api.h"

const int N = 1024;     // matrix size is N x N
//const int K = 32;       // tile size is K x K 

void transpose_cpu(int *in, int *out) 
{
        for (int row = 0; row < N; row++) {
                for (int col = 0; col < N; col++) {
                        out[col * N + row] = in[row * N + col];
                }
        }
}

__global__ void transpose_serial(int *in, int *out) 
{
        for (int row = 0; row < N; row++) {
                for (int col = 0; col < N; col++) {
                        out[col * N + row] = in[row * N + col];
                }
        }
}

void print_matrix(int *in) 
{
        for (int row = 0; row < N; row++) {
                for (int col = 0; col < N; col++) {
                        printf("%d ", in[row * N + col]);
                }
                printf("\n");
        }
}

void fill_matrix(int *in) 
{
        int size = N * N;
        for (int i = 0; i < size; i++) {
                in[i] = rand() % 10;
        }
}

// return (matrix a == matrix b)
bool same_matrices(int *a, int *b)
{
        int size = N * N;
        for (int i = 0; i < size; i++) {
                if (a[i] != b[i]) {
                        return false;
                }
        }
        return true;
}

int main(int argc, char **argv) 
{
        hipEvent_t start, stop;
        struct timeval start_time, stop_time;
        float elapsed_time;
        int num_bytes = N * N * sizeof(int);
        int *h_in = (int*)malloc(num_bytes);
        int *h_out = (int*)malloc(num_bytes);
        int *expected_out = (int*)malloc(num_bytes);
        int *d_in, *d_out;

        // no enough host memory
        if (!h_in || !h_out || !expected_out) {
                goto out;
        }

        // initialize matrix with random numbers
        fill_matrix(h_in);

        // transpose the matrix and get the expected matrix
        gettimeofday(&start_time, NULL);
        transpose_cpu(h_in, expected_out);
        gettimeofday(&stop_time, NULL);
        elapsed_time = (stop_time.tv_sec - start_time.tv_sec) * 1000 + (stop_time.tv_usec - start_time.tv_usec) / 1000.0;
        printf("CPU time: %f ms\n", elapsed_time);

        // allocate GPU memory
        checkCudaErrors(hipMalloc(&d_in, num_bytes));
        checkCudaErrors(hipMalloc(&d_out, num_bytes));

        // copy input from host memory to GPU memory
        checkCudaErrors(hipMemcpy(d_in, h_in, num_bytes, hipMemcpyHostToDevice));

        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        // launch serial kernel
        hipEventRecord(start);
        transpose_serial<<<1, 1>>>(d_in, d_out);
        hipEventRecord(stop);

        // copy output from GPU memory to host memory
        checkCudaErrors(hipMemcpy(h_out, d_out, num_bytes, hipMemcpyDeviceToHost));

        // calculate elapsed time in ms and check results
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        printf("transpose_serial time: %f ms\n%s results\n", elapsed_time, 
               same_matrices(h_out, expected_out) ? "Correct" : "Wrong");

        // free GPU memory
        hipFree(d_in);
        hipFree(d_out);

out:
        free(h_in);
        free(h_out);

        return 0;
}