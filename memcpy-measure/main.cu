
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sched.h>

void usage(char *program)
{     
        fprintf(stderr, "usage: %s memsize iters [-a]\n", program);
        fprintf(stderr, "    memsize: memory transferred in bytes (>0)\n");
        fprintf(stderr, "    iters  : number of iterations (>0)\n");
}

// copy size worth of bytes from h_in to d, then from d back to h_out for iters rounds
void profile_memcpy(char *h_in, char *h_out, char *d, int size, int iters)
{
        hipEvent_t start, stop;
        float time, throughput, avg_time;
        int i;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        // memory transfer from host to device
        hipEventRecord(start, 0);
        for (i = 0; i < iters; i++) {
                if (hipMemcpy(d, h_in, size, hipMemcpyHostToDevice) != hipSuccess) {
                        fprintf(stderr, "Error: memcpy from host to device\n");
                }
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        // calculate host to device information
        avg_time = time / iters / 1000;  // time in second
        throughput = (float)size / avg_time / 1000000000; // throughput in GB/s

        printf("  Host to Device Time: %.6f s\n", avg_time);
        printf("  Host to Device Throughput: %.6f GB/s\n", throughput);

        // memory transfer from device to host
        hipEventRecord(start, 0);        
        for (i = 0; i < iters; i++) {
                if (hipMemcpy(h_out, d, size, hipMemcpyDeviceToHost) != hipSuccess) {
                        fprintf(stderr, "Error: memcpy from device to host\n");
                }
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);        
        
        // calculate device to host information
        avg_time = time / iters / 1000;  // time in second
        throughput = (float)size / avg_time / 1000000000; // throughput in GB/s

        printf("  Device to Host Time: %.6f s\n", avg_time);
        printf("  Device to Host Throughput: %.6f GB/s\n", throughput);

        hipEventDestroy(start);
        hipEventDestroy(stop);
}

int main(int argc, char **argv)
{
        int size, iters;
        char *h_in_pageable, *h_out_pageable;   // host pageable memory 
        char *h_in_pinned, *h_out_pinned;       // host pinned memory 
        char *d;        // device memory

        if (argc != 3) {
                usage(argv[0]);
                return EXIT_FAILURE;
        }

        size = atoi(argv[1]);
        iters = atoi(argv[2]);

        if (size <= 0 || iters <= 0) {
                usage(argv[0]);
                return EXIT_FAILURE;
        }

        // allocate host pageable memory
        h_in_pageable = (char*)malloc(size);
        h_out_pageable = (char*)malloc(size);
        if (!h_in_pageable || !h_out_pageable) {
                fprintf(stderr, "Error: allocate host pageable memory\n");
                free(h_in_pageable);
                free(h_out_pageable);
                return EXIT_FAILURE;
        }

        // allocate host pinned memory
        if (hipHostMalloc((void**)&h_in_pinned, size, hipHostMallocDefault) != hipSuccess ||
            hipHostMalloc((void**)&h_out_pinned, size, hipHostMallocDefault) != hipSuccess) {
                fprintf(stderr, "Error: allocate host pinned memory\n");
                free(h_in_pageable);
                free(h_out_pageable);
                hipHostFree(h_in_pinned);
                hipHostFree(h_out_pinned);
                return EXIT_FAILURE;                                    
        } 

        // allocate device memory
        if (hipMalloc(&d, size) != hipSuccess) {
                fprintf(stderr, "Error: allocate device memory\n"); 
                free(h_in_pageable);
                free(h_out_pageable);
                hipHostFree(h_in_pinned);
                hipHostFree(h_out_pinned);   
                return EXIT_FAILURE;
        }

        // warm up
        hipFree(0);

        // Profile memory copy
        printf("Transfer size (MB): %f\n\n", (float)size / (1024 * 1024));
        printf("Pageable transfers\n");
        profile_memcpy(h_in_pageable, h_out_pageable, d, size, iters);
        printf("\n");
        printf("Pinned transfers\n");
        profile_memcpy(h_in_pinned, h_out_pinned, d, size, iters);

        // free memory
        free(h_in_pageable);
        free(h_out_pageable);
        hipHostFree(h_in_pinned);
        hipHostFree(h_out_pinned);   
        hipFree(d);

        return EXIT_SUCCESS;
}