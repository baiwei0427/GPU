
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void usage(char *program)
{     
        fprintf(stderr, "usage: %s memsize iters\n", program);
        fprintf(stderr, "    memsize: memory transferred in bytes (>0)\n");
        fprintf(stderr, "    iters: number of iterations (>0)\n");
}

int main(int argc, char **argv)
{
        int size, iters, i;
        void *h_ptr, *d_ptr;    // host memory, device memory
        hipEvent_t start, stop;
        float time, cumulative_h2d_time = 0.f, cumulative_d2h_time = 0.f, throughput, avg_time;

        if (argc != 3) {
                usage(argv[0]);
                return EXIT_FAILURE;
        }

        size = atoi(argv[1]);
        iters = atoi(argv[2]);

        if (size <= 0 || iters <= 0) {
                usage(argv[0]);
                return EXIT_FAILURE;
        }

        // allocate host memory
        if (!(h_ptr = malloc(size))) {
                fprintf(stderr, "Error: allocate host memory\n");
                return EXIT_FAILURE;
        }

        // allocate device memory
        if (hipMalloc(&d_ptr, size) != hipSuccess) {
                fprintf(stderr, "Error: allocate device memory\n"); 
                free(h_ptr);              
                return EXIT_FAILURE;
        }

        hipFree(0);
        hipEventCreate(&start);
        hipEventCreate(&stop);

        for (i = 0; i < iters; i++) {
                // memory transfer from host to device
                hipEventRecord(start, 0);
                if (hipMemcpy(d_ptr, h_ptr, size, hipMemcpyHostToDevice) != hipSuccess) {
                        fprintf(stderr, "Error: memcpy from host to device\n");
                }
                hipEventRecord(stop, 0);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&time, start, stop);
                cumulative_h2d_time += time;    

                // memory transfer from device to host
                hipEventRecord(start, 0);
                if (hipMemcpy(h_ptr, d_ptr, size, hipMemcpyDeviceToHost) != hipSuccess) {
                        fprintf(stderr, "Error: memcpy from device to host\n");
                }
                hipEventRecord(stop, 0);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&time, start, stop);
                cumulative_d2h_time += time;                    
        }

        free(h_ptr);
        hipFree(d_ptr);

        // calculate host to device information
        avg_time = cumulative_h2d_time / iters / 1000;  // time in second
        throughput = size / avg_time / 1000000; // throughput in MB/s
        
        printf("Host to Device Time: %.5f ms\n", avg_time);
        printf("Host to Device Throughput: %.5f MB/s\n", throughput);

        // calculate device to host information
        avg_time = cumulative_d2h_time / iters / 1000;  // time in second
        throughput = size / avg_time / 1000000; // throughput in MB/s
        
        printf("Device to Host Time: %.5f ms\n", avg_time);
        printf("Device to Host Throughput: %.5f MB/s\n", throughput);

        return EXIT_SUCCESS;
}