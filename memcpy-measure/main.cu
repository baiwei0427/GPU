
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sched.h>

void usage(char *program)
{     
        fprintf(stderr, "usage: %s memsize iters [-a]\n", program);
        fprintf(stderr, "    memsize: memory transferred in bytes (>0)\n");
        fprintf(stderr, "    iters  : number of iterations (>0)\n");
        fprintf(stderr, "    -a     : set CPU affinity\n");
}

// copy size worth of bytes from h_in to d, then from d back to h_out for iters rounds
void profile_memcpy(char *h_in, char *h_out, char *d, int size, int iters)
{
        hipEvent_t start, stop;
        float time, h2d_total_time, d2h_total_time, throughput, avg_time;
        int i;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        h2d_total_time = d2h_total_time = 0;

        for (i = 0; i < iters; i++) {
                // memory transfer from host to device
                hipEventRecord(start, 0);
                if (hipMemcpy(d, h_in, size, hipMemcpyHostToDevice) != hipSuccess) {
                        fprintf(stderr, "Error: memcpy from host to device\n");
                }
                hipEventRecord(stop, 0);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&time, start, stop);
                h2d_total_time += time;    

                // memory transfer from device to host
                hipEventRecord(start, 0);
                if (hipMemcpy(h_out, d, size, hipMemcpyDeviceToHost) != hipSuccess) {
                        fprintf(stderr, "Error: memcpy from device to host\n");
                }
                hipEventRecord(stop, 0);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&time, start, stop);
                d2h_total_time += time;     
        }

        // calculate host to device information
        avg_time = h2d_total_time / iters / 1000;  // time in second
        throughput = (float)size / avg_time / 1000000000; // throughput in GB/s

        printf("  Host to Device Time: %.6f s\n", avg_time);
        printf("  Host to Device Throughput: %.6f GB/s\n", throughput);

        // calculate device to host information
        avg_time = d2h_total_time / iters / 1000;  // time in second
        throughput = (float)size / avg_time / 1000000000; // throughput in GB/s

        printf("  Device to Host Time: %.6f s\n", avg_time);
        printf("  Device to Host Throughput: %.6f GB/s\n", throughput);

        hipEventDestroy(start);
        hipEventDestroy(stop);
}

int main(int argc, char **argv)
{
        int size, iters;
        char *h_in_pageable, *h_out_pageable;   // host pageable memory 
        char *h_in_pinned, *h_out_pinned;       // host pinned memory 
        char *d;        // device memory
        bool set_affinity = false;     
        cpu_set_t  mask;

        if (!(argc == 3 || (argc == 4 && strcmp(argv[3], "-a") == 0))) {
                usage(argv[0]);
                return EXIT_FAILURE;
        }

        size = atoi(argv[1]);
        iters = atoi(argv[2]);

        if (size <= 0 || iters <= 0) {
                usage(argv[0]);
                return EXIT_FAILURE;
        }

        if (argc == 4) {
                set_affinity = true;
        }

        // set a process's CPU affinity mask
        if (set_affinity) {
                CPU_ZERO(&mask);
                CPU_SET(0, &mask);
                if (sched_setaffinity(0, sizeof(mask), &mask) != 0) {
                        fprintf(stderr, "Error: set CPU affinity\n");
                        return EXIT_FAILURE;
                }
        }

        // allocate host pageable memory
        h_in_pageable = (char*)malloc(size);
        h_out_pageable = (char*)malloc(size);
        if (!h_in_pageable || !h_out_pageable) {
                fprintf(stderr, "Error: allocate host pageable memory\n");
                free(h_in_pageable);
                free(h_out_pageable);
                return EXIT_FAILURE;
        }

        // allocate host pinned memory
        if (hipHostMalloc((void**)&h_in_pinned, size, hipHostMallocDefault) != hipSuccess ||
            hipHostMalloc((void**)&h_out_pinned, size, hipHostMallocDefault) != hipSuccess) {
                fprintf(stderr, "Error: allocate host pinned memory\n");
                free(h_in_pageable);
                free(h_out_pageable);
                hipHostFree(h_in_pinned);
                hipHostFree(h_out_pinned);
                return EXIT_FAILURE;                                    
        } 

        // allocate device memory
        if (hipMalloc(&d, size) != hipSuccess) {
                fprintf(stderr, "Error: allocate device memory\n"); 
                free(h_in_pageable);
                free(h_out_pageable);
                hipHostFree(h_in_pinned);
                hipHostFree(h_out_pinned);   
                return EXIT_FAILURE;
        }

        // warm up
        hipFree(0);

        // Profile memory copy
        printf("Transfer size (MB): %f\n\n", (float)size / (1024 * 1024));
        printf("Pageable transfers\n");
        profile_memcpy(h_in_pageable, h_out_pageable, d, size, iters);
        printf("\n");
        printf("Pinned transfers\n");
        profile_memcpy(h_in_pinned, h_out_pinned, d, size, iters);

        // free memory
        free(h_in_pageable);
        free(h_out_pageable);
        hipHostFree(h_in_pinned);
        hipHostFree(h_out_pinned);   
        hipFree(d);

        return EXIT_SUCCESS;
}