#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__global__ void saxpy(int n, float a, float *x, float *y)
{
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < n) 
                y[i] = a * x[i] + y[i];
}

int main(void)
{
        int N = 1 << 20;
        float *x, *y, *d_x, *d_y;
        
        if (!(x = (float*)malloc(N * sizeof(float)))) {
                return 0;
        }

        if (!(y = (float*)malloc(N * sizeof(float)))) {
                return 0;
        }

        checkCudaErrors(hipMalloc(&d_x, N * sizeof(float))); 
        checkCudaErrors(hipMalloc(&d_y, N * sizeof(float)));

        for (int i = 0; i < N; i++) {
                x[i] = 1.0f;
                y[i] = 2.0f;
        }     

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        checkCudaErrors(hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice));

        hipEventRecord(start);
        // Perform SAXPY on 1M elements
        saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);
        hipEventRecord(stop);

        checkCudaErrors(hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost));
        
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        float maxError = 0.0f;
        for (int i = 0; i < N; i++)
                maxError = max(maxError, abs(y[i] - 4.0f));
        
        printf("Max error: %f\n", maxError);
        printf("Elapsed time: %f ms\n", milliseconds);

        hipFree(d_x);
        hipFree(d_y);
        free(x);
        free(y);
}